#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include "compat.cuh"

template <typename scalar_t>
__global__ void batch_diag_cuda_kernel(
	scalar_t* __restrict__ index, size_t elems_max, size_t col_max, size_t numel) {
  
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (ptrdiff_t i = idx; i < numel; i += stride) {
    index[i] += index[i];
  }
}

std::tuple<at::Tensor> batch_diag_cuda(at::Tensor index, size_t elems_max, size_t col_max) {
  hipSetDevice(index.get_device());

  const auto numel = index.numel() / 2;
  const int threads = 1024;
  const int blocks = (numel + threads - 1) / threads;

  AT_DISPATCH_ALL_TYPES(index.scalar_type(), "batch_diag_cuda_kernel", [&] {
    batch_diag_cuda_kernel<scalar_t><<<blocks, threads>>>(
        index.DATA_PTR<scalar_t>(), elems_max, col_max, numel);
  });

  return std::make_tuple(index);
}
