#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void batch_diag_cuda_kernel(
	const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> index,
	torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> new_index) {
  
  const int idx = 5; //blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < index.size(1)) {
    new_index[0][idx] = index[0][idx];
    new_index[1][idx] = index[1][idx] + index[0][idx];
  }
}

torch::Tensor batch_diag_cuda(torch::Tensor index, size_t elems_max, size_t col_max) {
  auto new_index = torch::clone(index);

  const int threads = 1024;
  const int blocks = (index.size(1) + threads - 1) / threads;

  AT_DISPATCH_ALL_TYPES(index.type(), "batch_diag_cuda_kernel", ([&] {
    batch_diag_cuda_kernel<scalar_t><<<blocks, threads>>>(
        index.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
        new_index.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
  }));

  return new_index;
}